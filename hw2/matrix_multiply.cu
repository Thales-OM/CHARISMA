#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cstdio>

// #include "matrix_multiply.cuh"


#define TILE_SIZE 16 // Размер блока (TILE_SIZE x TILE_SIZE)

// CUDA Kernel для умножения матриц
__global__ void squareMatrixMultiply(double *A, double *B, double *C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y; // Индекс строки
    int col = blockIdx.x * blockDim.x + threadIdx.x; // Индекс столбца

    // Проверка, что не вылетели за границы итоговой матрицы
    if (row < n && col < n) {
        double sum = 0.0;
        for (int k = 0; k < n; ++k) {
            sum += A[k * n + col] * B[row * n + k]; // Умножение с учетом column-major порядка
        }
        C[row * n + col] = sum;
    }
}

// Логика умножения матриц через CUDA kernell
extern "C" void multiplyMatrices(double *h_A, double *h_B, double *h_C, int n) {
    double *d_A, *d_B, *d_C;

    // Выделение памяти на GPU
    hipMalloc((void **)&d_A, n * n * sizeof(double));
    hipMalloc((void **)&d_B, n * n * sizeof(double));
    hipMalloc((void **)&d_C, n * n * sizeof(double));

    // Создание событий для измерения времени
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Запуск события перед выполнением ядра
    hipEventRecord(start);

    // Копирование матриц A и B на GPU
    hipMemcpy(d_A, h_A, n * n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, n * n * sizeof(double), hipMemcpyHostToDevice);

    // Определение размеров блоков и сетки
    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 numBlocks((n + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (n + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Вызов ядра
    squareMatrixMultiply<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, n);
    hipDeviceSynchronize();

    // Копирование результата обратно на CPU
    hipMemcpy(h_C, d_C, n * n * sizeof(double), hipMemcpyDeviceToHost);

    // Запуск события после выполнения ядра
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Измерение времени выполнения
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Time taken for matrix multiplication (CUDA): %.2f ms\n", elapsedTime);

    // Освобождение памяти
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

// Логика умножения матриц через CUDA kernell с помощью Pinned памяти
extern "C" void pinnedMultiplyMatrices(double *A, double *B, double *C, int n) {
    hipEvent_t start, stop;
    float elapsedTime;
    double *d_A, *d_B, *d_C;

    hipHostMalloc(&A, n * n * sizeof(double), hipHostMallocDefault);
    hipHostMalloc(&B, n * n * sizeof(double), hipHostMallocDefault);
    hipMalloc(&d_A, n * n * sizeof(double));
    hipMalloc(&d_B, n * n * sizeof(double));
    hipMalloc(&d_C, n * n * sizeof(double));

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    hipMemcpyAsync(d_A, A, n * n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_B, B, n * n * sizeof(double), hipMemcpyHostToDevice);

    int threadsPerBlock = TILE_SIZE;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    squareMatrixMultiply<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, n);
    // cudaDeviceSynchronize();
    
    hipMemcpyAsync(C, d_C, n * n * sizeof(double), hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    printf("Time taken for matrix multiplication (CUDA): %.2f ms\n", elapsedTime);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

// Type definition for the matrix generation function pointer
typedef void (*MatrixGenFunc)(double*, int);

// Логика умножения матриц через CUDA kernell с помощью Unified памяти
extern "C" void unifiedMultiplyMatrices(MatrixGenFunc matrixGenFunc, int n) {
    double *A, *B, *C;
    hipMallocManaged(&A, n * n * sizeof(double));
    hipMallocManaged(&B, n * n * sizeof(double));
    hipMallocManaged(&C, n * n * sizeof(double));

    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Генерация случайных матриц
    matrixGenFunc(A, n);
    matrixGenFunc(B, n);

    hipEventRecord(start);

    int threadsPerBlock = TILE_SIZE;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    squareMatrixMultiply<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, n);
    // cudaDeviceSynchronize();


    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    printf("Time taken for matrix multiplication (CUDA): %.2f ms\n", elapsedTime);

    hipFree(A);
    hipFree(B);
    hipFree(C);
}

// Логика умножения матриц через CUDA kernell с помощью CUDA-streams
extern "C" void streamingMultiplyMatrices(double *A, double *B, double *C, int n) {
    double *d_A, *d_B, *d_C;
    hipEvent_t start, stop;
    hipStream_t stream1, stream2, stream3, stream4;

    hipMalloc(&d_A, n * n * sizeof(double));
    hipMalloc(&d_B, n * n * sizeof(double));
    hipMalloc(&d_C, n * n * sizeof(double));
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);
    hipStreamCreate(&stream4);

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    hipMemcpyAsync(d_A, A, n * n * sizeof(double), hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(d_B, B, n * n * sizeof(double), hipMemcpyHostToDevice, stream2);

    int threadsPerBlock = TILE_SIZE;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    squareMatrixMultiply<<<blocksPerGrid, threadsPerBlock, 0, stream3>>>(d_A, d_B, d_C, n);

    hipMemcpyAsync(C, d_C, n * n * sizeof(double), hipMemcpyDeviceToHost, stream4);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    // Wait for the stream to complete
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);
    hipStreamSynchronize(stream3);
    hipStreamSynchronize(stream4);
    
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    printf("Time taken for matrix multiplication (CUDA): %.2f ms\n", elapsedTime);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipStreamDestroy(stream3);
    hipStreamDestroy(stream4);
}